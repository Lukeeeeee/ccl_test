


#include "hip/hip_runtime.h"


#include "device_atomic_functions.h"

#define CUDA_CALL(x) {const hipError_t a = (x);if (a != hipSuccess) { printf("\nCuda error: %s (err_num = %d)\n",hipGetErrorString(a),a);hipDeviceReset();}}
#define MAX_HEIGHT 1024
#define MAX_WIDTH 1024
//#define MAX_VERTEX MAX_WIDTH * MAX_HEIGHT

#define INT_PTR(x) (*((int*)(&(x))))

__device__ int dx[8] = {-1,0,1,-1,1,-1,0,1};
__device__ int dy[8] = {-1,-1,-1,0,0,1,1,1};

__device__ int get_loc() {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	return gridDim.x * blockDim.x * idy +idx;
} 

__device__ int get_x() {
	return blockIdx.x * blockDim.x + threadIdx.x;
}

__device__ int get_y() {
	return blockIdx.y * blockDim.y + threadIdx.y;
}

__device__ bool check_bound(int x, int y, int w, int h) {
	if (x>0&&y>0&&x<w&&y<h) return 1;
		else return 0;
}

__device__ bool check_connect(int loc1, int loc2, unsigned char *img, unsigned char byF) {
	if(img[loc1]==byF&&img[loc2]==byF) return 1;
	return 0;





}